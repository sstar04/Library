#include "hip/hip_runtime.h"
/*
 * TMEInspector.cu
 *
 *  Created on: 25/mag/2018
 *      Author: Sabrina
 */

#include <math.h> 
#include <stdio.h>
#include <algorithm>    
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include "TMEInspector.h"
#include "TMEInspector_kernel.cu"
#include "Filter.h"


TMEInspector::TMEInspector(){
	// TODO Auto-generated constructor stub

}

TMEInspector::TMEInspector(TGSimulator& tgSim): m_tgSim(tgSim){

   // Allocate Unified Memory -- accessible from CPU or GPU
   //printf("numero totale di cellule %d", m_tgSim.Get_numCells() );
      hipMallocManaged(&m_border, m_tgSim.Get_numCells()*sizeof(bool));
      hipDeviceSynchronize();

      //initialize Filter instance
      m_filter = Filter(m_tgSim.Get_VCells(),m_tgSim.Get_numCells());
      printf("Sono in TMEInspector \n");
}

TMEInspector::~TMEInspector() {
	
	hipFree(m_border);
}


bool* TMEInspector::Search_BorderCells(){


	unsigned int numCells = m_tgSim.Get_numCells();

	// Launch kernel on numCells elements on the GPU
	int blockSize = 16;
	int numBlocks = ceil(numCells/blockSize)+1;
	printf("numero di blocchi %d - numeri di threads: %d", numBlocks, blockSize);
	//   init<<<1, numCells>>>(numCells, m_border);
	init<<<numBlocks, blockSize>>>(numCells, m_border);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	return m_border;
}


bool* TMEInspector::Search_BorderCellsTh(){


    unsigned int numCells = m_tgSim.Get_numCells();

    //inizializzo m_border con trust utilizzando un wrapper
    
    // wrap raw pointer with a device_ptr 
    thrust::device_ptr<bool> dev_border = thrust::device_pointer_cast(m_border);

    // use device_ptr in Thrust algorithms
    thrust::fill(dev_border, dev_border + numCells , 1);
  
    //init<<<numBlocks, blockSize>>>(numCells, m_border);   // !non funziona
    
    //Wait for GPU to finish before accessing on host  
    hipDeviceSynchronize();
      
   return m_border;
}



void TMEInspector::Search_VecBorderCells(std::vector<bool>& IsonAS){

	//FindBorder(). In attesa di implementare questa funzione privata mi limito semplicemente a inizializzare il vettore

	unsigned int numCells = m_tgSim.Get_numCells();
	// Launch kernel on numCells elements on the GPU
	int blockSize = 16;
	int numBlocks = ceil(numCells/blockSize)+1;
	init<<<numBlocks, blockSize>>>(numCells, m_border);

	//Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	std::copy(m_border, m_border+numCells, IsonAS.begin());
}


void TMEInspector::STARAlgorithm(std::vector<bool>& IsonAS,
									float a )//! algorithm's runtime parameter (USide= a*r_max)
{

	  //Calculate Extended bounding box
	  float U_side = (a*m_tgSim.Get_rmax());


#ifdef DEBUG
	  std::cout << "**** TMEInspector::STARAlgorithm says:" << std::endl;
	  std::cout << "runtime parameter 'a' of STARAlgorithm is set to: " << a << std::endl;
	  std::cout << "the maximum value of the cell's radius is: " << m_tgSim.Get_rmax() << std::endl;
	  std::cout << "the side of the small cubic box is: " <<a*m_tgSim.Get_rmax() << "micron" << std::endl<< std::endl;
#endif

	  m_filter.Calculate_ExtendedBoundingBox(U_side,2);

	  //m_filter.TestNewData();
	  m_filter.ReorderCellsData_by_HashValue();

	return;

}
